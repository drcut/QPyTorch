#include "hip/hip_runtime.h"
#include "quant_kernel.h"
#include "bit_helper.cu"
#include <cstdio>
#include <hip/hip_fp16.h>

// quantize a float into a floating point with [exp_bits] exponent and
// [man_bits] mantissa
__global__ void float_kernel_stochastic(float* __restrict__ a,
                                        int* __restrict__ r,
                                        float* o, int size,
                                        int man_bits,
                                        int exp_bits) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    unsigned int old_num = FLOAT_TO_BITS(&a[index]);
    unsigned int rand_prob = (unsigned int) r[index];
    unsigned int quantize = round_bitwise_stochastic(old_num, rand_prob, man_bits);
    quantize = clip_exponent(exp_bits, man_bits, old_num, quantize);
    float quantize_float = BITS_TO_FLOAT(&quantize);
    o[index] = quantize_float;
  }
}

// quantize a float into a floating point with [exp_bits] exponent and
// [man_bits] mantissa
__global__ void float_kernel_nearest(float* __restrict__ a,
                                     float* o, int size,
                                     int man_bits,
                                     int exp_bits) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    unsigned int old_num = FLOAT_TO_BITS(&a[index]);
    unsigned int exp = (old_num & 0x7F800000)>>23;
    unsigned int man = (old_num & 0x007FFFFF);
    int true_exp = (int)exp - 127;
    if(exp > 0) { // normal float 
       man = man | (1<<23); 
       const int DIY_bias = (1<<(exp_bits-1)) - 1;
       int new_e = true_exp + DIY_bias;
       if(new_e > 0) { // normal number for DIY precision
          // round man
          if((man & 1<<(23 - man_bits - 1)) == 0) // just round to lower
          {
             man = man & ~((1<<(23 - man_bits)) - 1);
          }
          else
          {
             if((man & ((1<<(23 - man_bits - 1)) -1)) != 0) { // just round to upper
                man = man + (1<<(23 - man_bits - 1));
                // check if the high position is changed
                if((man & (1<<(23+1))) == 0) // have not changed
                {
                    man = man & (~((1<<(23 - man_bits)) - 1));
                }
                else {
                    man = man >> 1;
                    man = man & ~((1<<(23 - man_bits)) - 1);
                    new_e += 1;
                }
             }
             else { // round to nearest even
                if(man & (1<<(23 - man_bits)) == 0) // just truncation
                    man = man & ~((1<<(23 - man_bits)) - 1);
                else {
                    man = man + (1<<(23 - man_bits - 1));
                    // check if the high position is changed
                    if(man & (1<<(23+1)) == 0) // have not changed
                    {
                        man = man & ~((1<<(23 - man_bits)) - 1);
                    }
                    else {
                        man = man >> 1;
                        man = man & ~((1<<(23 - man_bits)) - 1);
                        new_e += 1;
                    }
                }
             }
          }
          new_e -= DIY_bias;
       }
       else { //subnormal number for DIY precision
          man = man>>(-new_e);
          new_e = - DIY_bias;
          //TODO: add round part
       }
       // TODO: maybe e will be 0xFFF, this will cause bug
       if(new_e >=0)
            o[index] = ((float)man)/(1<<23) * (1<<new_e);
       else
            o[index] = ((float)man)/(1<<23) / (1<<(-new_e));
       if(a[index] < 0)
          o[index] = -o[index];
    }
  }
}
